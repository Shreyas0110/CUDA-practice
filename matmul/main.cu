#include "matmul.cu"
#include <fstream>
#include <iostream>

#define WIDTH 256
#define SIZE (WIDTH*WIDTH)

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float *a, *b, *out, *dev_a, *dev_b, *dev_out;
    a = (float *) malloc(SIZE * sizeof(float));
    b = (float *) malloc(SIZE * sizeof(float));
    out = (float *) malloc(SIZE * sizeof(float));

    for (int i = 0; i < SIZE; ++i){
        a[i] = i;
        b[i] = i;
    }

    hipMalloc((void **) &dev_a, SIZE * sizeof(float));
    hipMalloc((void **) &dev_b, SIZE * sizeof(float));
    hipMalloc((void **) &dev_out, SIZE * sizeof(float));

    hipMemcpy(dev_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, SIZE * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32, 1);
    int gridSize = (WIDTH+31)/32; 
    std::cout << gridSize << std::endl;
    dim3 dimGrid(gridSize, gridSize, 1);

    naiveMatMulKernel <<<dimGrid, dimBlock>>> (dev_a, dev_b, dev_out, WIDTH);

    hipMemcpy(out, dev_out, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_out);
    free(a);
    free(b);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);

    std::cout << "Time Taken: " << time << std::endl;

    std::fstream outfile;
    outfile.open("mymatmul.txt", std::fstream::out);
    for (int i = 0; i < SIZE; i++){
        if (i % WIDTH == 0 && i != 0)
            outfile << "\n";
        outfile << out[i] << " ";
    }
    outfile.close();
    free(out);
}