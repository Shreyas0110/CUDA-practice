
#include <hip/hip_runtime.h>

/* Assume Square Matix */
__global__ void naiveMatMulKernel(float* A, float* B, float* out, int width){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < width && col < width){
        float val = 0;
        for (int i = 0; i < width; i++){
            val += A[row * width + i] * B[width * i + col];
        }
        out[row * width + col] = val;
    }

}