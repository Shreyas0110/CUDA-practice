
#include <hip/hip_runtime.h>

/* Assume Square Matix */
__global__ void naiveMatMulKernel(float* A, float* B, float* out, int width){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < width && col < width){
        float val = 0;
        for (int i = 0; i < width; i++){
            val += A[row * width + i] * B[width * i + col];
        }
        out[row * width + col] = val;
    }
}


/* Think Tile Width must be same as sqrt of threads in block */
#define TILE_WIDTH 32
__global__ void tilingMatMulKernel(float* A, float* B, float* out, int width){

    __shared__ float A_shared [TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_shared [TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    float val = 0;

    for (int tile_index = 0; tile_index < width/TILE_WIDTH; tile_index++){
        A_shared [ty][tx] = A [row * width + tile_index*TILE_WIDTH + tx];
        B_shared [ty][tx] = B [(tile_index*TILE_WIDTH + ty)*width + col];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++){
            val += A_shared[ty][i] * B_shared[i][tx]; 
        }
        __syncthreads();
    }
    out[row * width + col] = val;
}